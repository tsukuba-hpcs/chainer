#include "hip/hip_runtime.h"
#include "xchainer/cuda/cuda_device.h"

#include <algorithm>
#include <memory>
#include <tuple>
#include <utility>

#include <hip/hip_runtime.h>

#include "xchainer/array.h"
#include "xchainer/cuda/hip/hip_runtime.h"
#include "xchainer/device.h"
#include "xchainer/dtype.h"
#include "xchainer/indexable_array.h"
#include "xchainer/indexer.h"
#include "xchainer/native_device.h"
#include "xchainer/scalar.h"

namespace xchainer {
namespace cuda {

namespace {

template <typename T>
__global__ void FillKernel(IndexableArray<T> out_iarray, T value, Indexer<> indexer) {
    for (int64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < indexer.total_size(); i += blockDim.x * gridDim.x) {
        indexer.Set(i);
        out_iarray[indexer] = value;
    }
}

template <typename T>
__global__ void AddKernel(IndexableArray<const T> lhs_iarray, IndexableArray<const T> rhs_iarray, IndexableArray<T> out_iarray,
                          Indexer<> indexer) {
    const int64_t total_size = indexer.total_size();
    for (int64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < total_size; i += blockDim.x * gridDim.x) {
        indexer.Set(i);
        out_iarray[indexer] = lhs_iarray[indexer] + rhs_iarray[indexer];
    }
}

template <typename T>
__global__ void MulKernel(IndexableArray<const T> lhs_iarray, IndexableArray<const T> rhs_iarray, IndexableArray<T> out_iarray,
                          Indexer<> indexer) {
    const int64_t total_size = indexer.total_size();
    for (int64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < total_size; i += blockDim.x * gridDim.x) {
        indexer.Set(i);
        out_iarray[indexer] = lhs_iarray[indexer] * rhs_iarray[indexer];
    }
}

}  // namespace

std::shared_ptr<void> CudaDevice::Allocate(size_t bytesize) {
    if (bytesize == 0) {
        return nullptr;
    }
    CheckError(hipSetDevice(index()));
    void* raw_ptr = nullptr;
    // Be careful to be exception-safe, i.e.,
    // do not throw any exceptions before creating shared_ptr when memory allocation is succeeded
    hipError_t status = hipMallocManaged(&raw_ptr, bytesize, hipMemAttachGlobal);
    if (status != hipSuccess) {
        cuda::Throw(status);
    }
    return std::shared_ptr<void>{raw_ptr, hipFree};
}

void CudaDevice::MemoryCopyFrom(void* dst, const void* src, size_t bytesize, Device& src_device) {
    assert(IsPointerCudaMemory(dst));
    if (&src_device == this || nullptr != dynamic_cast<CudaDevice*>(&src_device)) {
        // Copy between CUDA devices
        CheckError(hipMemcpy(dst, src, bytesize, hipMemcpyDeviceToDevice));
    } else {
        assert(nullptr != dynamic_cast<NativeDevice*>(&src_device) && "CudaDevice only supports copy between cuda or native devices.");
        // Copy from native device
        CheckError(hipMemcpy(dst, src, bytesize, hipMemcpyHostToDevice));
    }
}

void CudaDevice::MemoryCopyTo(void* dst, const void* src, size_t bytesize, Device& dst_device) {
    assert(IsPointerCudaMemory(src));
    if (&dst_device == this || nullptr != dynamic_cast<CudaDevice*>(&dst_device)) {
        // Copy between CUDA devices
        CheckError(hipMemcpy(dst, src, bytesize, hipMemcpyDeviceToDevice));
    } else {
        assert(nullptr != dynamic_cast<NativeDevice*>(&dst_device) && "CudaDevice only supports copy between cuda or native devices.");
        // Copy to native device
        CheckError(hipMemcpy(dst, src, bytesize, hipMemcpyDeviceToHost));
    }
}

std::tuple<std::shared_ptr<void>, size_t> CudaDevice::TransferDataFrom(Device& src_device, const std::shared_ptr<void>& src_ptr,
                                                                       size_t offset, size_t bytesize) {
    std::shared_ptr<void> dst_ptr = Allocate(bytesize);
    MemoryCopyFrom(dst_ptr.get(), &static_cast<int8_t*>(src_ptr.get())[offset], bytesize, src_device);
    return std::make_tuple(std::move(dst_ptr), 0);
}

std::tuple<std::shared_ptr<void>, size_t> CudaDevice::TransferDataTo(Device& dst_device, const std::shared_ptr<void>& src_ptr,
                                                                     size_t offset, size_t bytesize) {
    std::shared_ptr<void> dst_ptr = dst_device.Allocate(bytesize);
    MemoryCopyTo(dst_ptr.get(), &static_cast<int8_t*>(src_ptr.get())[offset], bytesize, dst_device);
    return std::make_tuple(std::move(dst_ptr), 0);
}

std::shared_ptr<void> CudaDevice::FromBuffer(const std::shared_ptr<void>& src_ptr, size_t bytesize) {
    std::shared_ptr<void> dst_ptr = Allocate(bytesize);
    CheckError(hipMemcpy(dst_ptr.get(), src_ptr.get(), bytesize, hipMemcpyHostToDevice));
    return dst_ptr;
}

void CudaDevice::Fill(Array& out, Scalar value) {
    VisitDtype(out.dtype(), [&](auto pt) {
        using T = typename decltype(pt)::type;
        static const int kMaxBlockSize = CudaOccupancyMaxPotentialBlockSize(&FillKernel<T>).block_size;

        IndexableArray<T> out_iarray{out};
        Indexer<> indexer{out.shape()};
        int64_t grid_size = (indexer.total_size() + kMaxBlockSize - 1) / kMaxBlockSize;
        int64_t block_size = std::min<int64_t>(indexer.total_size(), kMaxBlockSize);

        FillKernel<<<grid_size, block_size>>>(out_iarray, static_cast<T>(value), indexer);
    });
}

// TODO(sonots): support stream
void CudaDevice::Add(const Array& lhs, const Array& rhs, Array& out) {
    CheckDevicesCompatible(lhs, rhs, out);
    hipSetDevice(index());
    VisitDtype(lhs.dtype(), [&](auto pt) {
        using T = typename decltype(pt)::type;
        static const int kMaxBlockSize = CudaOccupancyMaxPotentialBlockSize(&AddKernel<T>).block_size;

        IndexableArray<const T> lhs_iarray{lhs};
        IndexableArray<const T> rhs_iarray{rhs};
        IndexableArray<T> out_iarray{out};
        Indexer<> indexer{lhs.shape()};

        int64_t total_size = indexer.total_size();
        int64_t grid_size = (total_size + kMaxBlockSize - 1) / kMaxBlockSize;
        int64_t block_size = std::min<int64_t>(total_size, kMaxBlockSize);

        AddKernel<<<grid_size, block_size>>>(lhs_iarray, rhs_iarray, out_iarray, indexer);
    });
}

// TODO(sonots): support stream
void CudaDevice::Mul(const Array& lhs, const Array& rhs, Array& out) {
    CheckDevicesCompatible(lhs, rhs, out);
    hipSetDevice(index());
    VisitDtype(lhs.dtype(), [&](auto pt) {
        using T = typename decltype(pt)::type;
        static const int kMaxBlockSize = CudaOccupancyMaxPotentialBlockSize(&MulKernel<T>).block_size;

        IndexableArray<const T> lhs_iarray{lhs};
        IndexableArray<const T> rhs_iarray{rhs};
        IndexableArray<T> out_iarray{out};
        Indexer<> indexer{lhs.shape()};

        int64_t total_size = indexer.total_size();
        int64_t grid_size = (total_size + kMaxBlockSize - 1) / kMaxBlockSize;
        int64_t block_size = std::min<int64_t>(total_size, kMaxBlockSize);

        MulKernel<<<grid_size, block_size>>>(lhs_iarray, rhs_iarray, out_iarray, indexer);
    });
}

void CudaDevice::Synchronize() {
    CheckError(hipSetDevice(index()));
    CheckError(hipDeviceSynchronize());
}

}  // namespace cuda
}  // namespace xchainer
