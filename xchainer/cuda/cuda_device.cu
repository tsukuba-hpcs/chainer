#include "hip/hip_runtime.h"
#include "xchainer/cuda/cuda_device.h"

#include <algorithm>
#include <memory>
#include <tuple>
#include <utility>

#include <hip/hip_runtime.h>

#include "xchainer/array.h"
#include "xchainer/cuda/hip/hip_runtime.h"
#include "xchainer/device.h"
#include "xchainer/dtype.h"
#include "xchainer/memory.h"
#include "xchainer/native_device.h"
#include "xchainer/scalar.h"

namespace xchainer {
namespace cuda {

namespace {

template <typename T>
__global__ void FillKernel(T* odata, T value, int64_t total_size) {
    for (int64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < total_size; i += blockDim.x * gridDim.x) {
        odata[i] = value;
    }
}

template <typename T>
__global__ void AddKernel(const T* ldata, const T* rdata, T* odata, int64_t total_size) {
    for (int64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < total_size; i += blockDim.x * gridDim.x) {
        odata[i] = ldata[i] + rdata[i];
    }
}

template <typename T>
__global__ void MulKernel(const T* ldata, const T* rdata, T* odata, int64_t total_size) {
    for (int64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < total_size; i += blockDim.x * gridDim.x) {
        odata[i] = ldata[i] * rdata[i];
    }
}

}  // namespace

std::shared_ptr<void> CudaDevice::Allocate(size_t bytesize) {
    if (bytesize == 0) {
        return nullptr;
    }
    CheckError(hipSetDevice(index()));
    void* raw_ptr = nullptr;
    // Be careful to be exception-safe, i.e.,
    // do not throw any exceptions before creating shared_ptr when memory allocation is succeeded
    hipError_t status = hipMallocManaged(&raw_ptr, bytesize, hipMemAttachGlobal);
    if (status != hipSuccess) {
        cuda::Throw(status);
    }
    return std::shared_ptr<void>{raw_ptr, hipFree};
}

void CudaDevice::MemoryCopyFrom(void* dst, const void* src, size_t bytesize, Device& src_device) {
    assert(internal::IsPointerCudaMemory(dst));
    if (&src_device == this || nullptr != dynamic_cast<CudaDevice*>(&src_device)) {
        // Copy between CUDA devices
        CheckError(hipMemcpy(dst, src, bytesize, hipMemcpyDeviceToDevice));
    } else {
        assert(nullptr != dynamic_cast<NativeDevice*>(&src_device) && "CudaDevice only supports copy between cuda or native devices.");
        // Copy from native device
        CheckError(hipMemcpy(dst, src, bytesize, hipMemcpyHostToDevice));
    }
}

void CudaDevice::MemoryCopyTo(void* dst, const void* src, size_t bytesize, Device& dst_device) {
    assert(internal::IsPointerCudaMemory(src));
    if (&dst_device == this || nullptr != dynamic_cast<CudaDevice*>(&dst_device)) {
        // Copy between CUDA devices
        CheckError(hipMemcpy(dst, src, bytesize, hipMemcpyDeviceToDevice));
    } else {
        assert(nullptr != dynamic_cast<NativeDevice*>(&dst_device) && "CudaDevice only supports copy between cuda or native devices.");
        // Copy to native device
        CheckError(hipMemcpy(dst, src, bytesize, hipMemcpyDeviceToHost));
    }
}

std::tuple<std::shared_ptr<void>, size_t> CudaDevice::TransferDataFrom(Device& src_device, const std::shared_ptr<void>& src_ptr,
                                                                       size_t offset, size_t bytesize) {
    std::shared_ptr<void> dst_ptr = Allocate(bytesize);
    MemoryCopyFrom(dst_ptr.get(), &static_cast<int8_t*>(src_ptr.get())[offset], bytesize, src_device);
    return std::make_tuple(std::move(dst_ptr), 0);
}

std::tuple<std::shared_ptr<void>, size_t> CudaDevice::TransferDataTo(Device& dst_device, const std::shared_ptr<void>& src_ptr,
                                                                     size_t offset, size_t bytesize) {
    std::shared_ptr<void> dst_ptr = dst_device.Allocate(bytesize);
    MemoryCopyTo(dst_ptr.get(), &static_cast<int8_t*>(src_ptr.get())[offset], bytesize, dst_device);
    return std::make_tuple(std::move(dst_ptr), 0);
}

std::shared_ptr<void> CudaDevice::FromBuffer(const std::shared_ptr<void>& src_ptr, size_t bytesize) {
    std::shared_ptr<void> dst_ptr = Allocate(bytesize);
    CheckError(hipMemcpy(dst_ptr.get(), src_ptr.get(), bytesize, hipMemcpyHostToDevice));
    return dst_ptr;
}

void CudaDevice::Fill(Array& out, Scalar value) {
    VisitDtype(out.dtype(), [&](auto pt) {
        using T = typename decltype(pt)::type;
        static const int kMaxBlockSize = CudaOccupancyMaxPotentialBlockSize(&FillKernel<T>).block_size;

        int64_t total_size = out.GetTotalSize();
        int64_t grid_size = (total_size + kMaxBlockSize - 1) / kMaxBlockSize;
        int64_t block_size = std::min<int64_t>(total_size, kMaxBlockSize);

        auto* odata = static_cast<T*>(out.data().get());
        FillKernel<<<grid_size, block_size>>>(odata, static_cast<T>(value), total_size);
    });
}

// TODO(sonots): support stream
void CudaDevice::Add(const Array& lhs, const Array& rhs, Array& out) {
    CheckDevicesCompatible(lhs, rhs, out);
    hipSetDevice(index());
    VisitDtype(lhs.dtype(), [&](auto pt) {
        using T = typename decltype(pt)::type;
        static const int kMaxBlockSize = CudaOccupancyMaxPotentialBlockSize(&AddKernel<T>).block_size;

        int64_t total_size = lhs.GetTotalSize();
        int64_t grid_size = (total_size + kMaxBlockSize - 1) / kMaxBlockSize;
        int64_t block_size = std::min<int64_t>(total_size, kMaxBlockSize);

        auto* ldata = static_cast<const T*>(lhs.data().get());
        auto* rdata = static_cast<const T*>(rhs.data().get());
        auto* odata = static_cast<T*>(out.data().get());
        AddKernel<<<grid_size, block_size>>>(ldata, rdata, odata, total_size);
    });
}

// TODO(sonots): support stream
void CudaDevice::Mul(const Array& lhs, const Array& rhs, Array& out) {
    CheckDevicesCompatible(lhs, rhs, out);
    hipSetDevice(index());
    VisitDtype(lhs.dtype(), [&](auto pt) {
        using T = typename decltype(pt)::type;
        static const int kMaxBlockSize = CudaOccupancyMaxPotentialBlockSize(&MulKernel<T>).block_size;

        int64_t total_size = lhs.GetTotalSize();
        int64_t grid_size = (total_size + kMaxBlockSize - 1) / kMaxBlockSize;
        int64_t block_size = std::min<int64_t>(total_size, kMaxBlockSize);

        auto* ldata = static_cast<const T*>(lhs.data().get());
        auto* rdata = static_cast<const T*>(rhs.data().get());
        auto* odata = static_cast<T*>(out.data().get());
        MulKernel<<<grid_size, block_size>>>(ldata, rdata, odata, total_size);
    });
}

void CudaDevice::Synchronize() {
    CheckError(hipSetDevice(index()));
    CheckError(hipDeviceSynchronize());
}

}  // namespace cuda
}  // namespace xchainer
