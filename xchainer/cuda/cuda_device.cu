#include "xchainer/cuda/cuda_device.h"

#include <hipblas.h>

#include <algorithm>
#include <cassert>
#include <cmath>
#include <cstddef>
#include <cstdint>
#include <memory>
#include <numeric>

#include <hip/hip_runtime.h>

#include "xchainer/array.h"
#include "xchainer/axes.h"
#include "xchainer/backend_util.h"
#include "xchainer/cuda/cast.cuh"
#include "xchainer/cuda/hipblas.h"
#include "xchainer/cuda/hip/hip_runtime.h"
#include "xchainer/cuda/hipDNN.h"
#include "xchainer/cuda/elementwise.cuh"
#include "xchainer/device.h"
#include "xchainer/dtype.h"
#include "xchainer/enum.h"
#include "xchainer/error.h"
#include "xchainer/indexable_array.h"
#include "xchainer/indexer.h"
#include "xchainer/native/native_device.h"
#include "xchainer/routines/connection.h"
#include "xchainer/routines/creation.h"
#include "xchainer/scalar.h"
#include "xchainer/shape.h"

namespace xchainer {
namespace cuda {

CudaDevice::~CudaDevice() {
    if (cublas_handle_) {
        hipSetDevice(index());
        hipblasDestroy(cublas_handle_);
    }
}

hipblasHandle_t CudaDevice::cublas_handle() {
    if (!cublas_handle_) {
        CheckCudaError(hipSetDevice(index()));
        CheckCublasError(hipblasCreate(&cublas_handle_));
    }
    return cublas_handle_;
}

std::shared_ptr<void> CudaDevice::Allocate(size_t bytesize) {
    CheckCudaError(hipSetDevice(index()));
    void* ptr = memory_pool_.Malloc(bytesize);
    return std::shared_ptr<void>{ptr, [this](void* ptr) { memory_pool_.Free(ptr); }};
}

std::shared_ptr<void> CudaDevice::MakeDataFromForeignPointer(const std::shared_ptr<void>& data) {
    // check memory validity
    void* ptr = data.get();
    hipPointerAttribute_t attr{};
    hipError_t status = hipPointerGetAttributes(&attr, ptr);
    switch (status) {
        case hipSuccess:
            if (attr.isManaged == 0) {
                throw XchainerError{"CUDA memory: ", ptr, " must be a managed (unified) memory"};
            }
            if (attr.device != index()) {
                throw XchainerError{"CUDA memory: ", ptr, " must reside on the device: ", index()};
            }
            break;
        case hipErrorInvalidValue:
            throw XchainerError{"Memory: ", ptr, " is not a CUDA memory"};
        default:
            Throw(status);
    }
    return data;
}

void CudaDevice::MemoryCopyFrom(void* dst, const void* src, size_t bytesize, Device& src_device) {
    assert(bytesize == 0 || IsPointerCudaMemory(dst));
    if (bytesize == 0) {
        return;
    }
    CheckCudaError(hipSetDevice(index()));
    if (&src_device == this || nullptr != dynamic_cast<CudaDevice*>(&src_device)) {
        // Copy between CUDA devices
        CheckCudaError(hipMemcpy(dst, src, bytesize, hipMemcpyDeviceToDevice));
    } else {
        assert(nullptr != dynamic_cast<native::NativeDevice*>(&src_device) &&
               "CudaDevice only supports copy between cuda or native devices.");
        // Copy from native device
        CheckCudaError(hipMemcpy(dst, src, bytesize, hipMemcpyHostToDevice));
    }
}

void CudaDevice::MemoryCopyTo(void* dst, const void* src, size_t bytesize, Device& dst_device) {
    assert(bytesize == 0 || src == nullptr || IsPointerCudaMemory(src));
    if (bytesize == 0) {
        return;
    }
    CheckCudaError(hipSetDevice(index()));
    if (&dst_device == this || nullptr != dynamic_cast<CudaDevice*>(&dst_device)) {
        // Copy between CUDA devices
        CheckCudaError(hipMemcpy(dst, src, bytesize, hipMemcpyDeviceToDevice));
    } else {
        assert(nullptr != dynamic_cast<native::NativeDevice*>(&dst_device) &&
               "CudaDevice only supports copy between cuda or native devices.");
        // Copy to native device
        CheckCudaError(hipMemcpy(dst, src, bytesize, hipMemcpyDeviceToHost));
    }
}

std::shared_ptr<void> CudaDevice::TransferDataFrom(
        Device& src_device, const std::shared_ptr<void>& src_ptr, size_t offset, size_t bytesize) {
    std::shared_ptr<void> dst_ptr = Allocate(bytesize);
    MemoryCopyFrom(dst_ptr.get(), &(static_cast<int8_t*>(src_ptr.get())[offset]), bytesize, src_device);
    return dst_ptr;
}

std::shared_ptr<void> CudaDevice::TransferDataTo(Device& dst_device, const std::shared_ptr<void>& src_ptr, size_t offset, size_t bytesize) {
    std::shared_ptr<void> dst_ptr = dst_device.Allocate(bytesize);
    MemoryCopyTo(dst_ptr.get(), &(static_cast<int8_t*>(src_ptr.get())[offset]), bytesize, dst_device);
    return dst_ptr;
}

std::shared_ptr<void> CudaDevice::FromHostMemory(const std::shared_ptr<void>& src_ptr, size_t bytesize) {
    std::shared_ptr<void> dst_ptr = Allocate(bytesize);
    CheckCudaError(hipMemcpy(dst_ptr.get(), src_ptr.get(), bytesize, hipMemcpyHostToDevice));
    return dst_ptr;
}

namespace {

void ConvCheckDtype(const Array& x, const Array& w, const nonstd::optional<Array>& b) {
    // TODO(sonots): Support float16
    if (x.dtype() != Dtype::kFloat32 && x.dtype() != Dtype::kFloat64) {
        throw XchainerError{"XChainer cuDNN supports only float32 or float64 arrays, but the input array dtype is: ", x.dtype()};
    }
    if (w.dtype() != x.dtype()) {
        throw XchainerError{"XChainer cuDNN requires the filter (kernel) array dtype: ",
                            w.dtype(),
                            " and the input array dtype: ",
                            x.dtype(),
                            " are same"};
    }
    if (b && b->dtype() != x.dtype()) {
        throw XchainerError{
                "XChainer cuDNN requires the bias array dtype: ", b->dtype(), " and the input array dtype: ", x.dtype(), " are same"};
    }
}

}  // namespace

Array CudaDevice::Conv(
        const Array& x,
        const Array& w,
        const nonstd::optional<Array>& b,
        const StackVector<int64_t, kMaxNdim>& stride,
        const StackVector<int64_t, kMaxNdim>& pad,
        bool cover_all) {
    if (cover_all) {
        throw XchainerError{"CUDA convolution does not support cover_all"};
    }
    if (b) {
        CheckDevicesCompatible(x, w, *b);
    } else {
        CheckDevicesCompatible(x, w);
    }
    ConvCheckDtype(x, w, b);

    int8_t ndim = x.ndim() - 2;  // Number of spacial dimensions
    if (ndim < 2) {
        throw DimensionError{"CUDA convolution requires number of spacial dimensions to be greater than or equal to 2"};
    }
    assert(w.ndim() == x.ndim());
    assert(stride.size() == static_cast<size_t>(ndim));
    assert(pad.size() == static_cast<size_t>(ndim));

    // w.shape = (out_channels, _, k_1, k_2, ..., k_N)
    int64_t out_channels = w.shape()[0];
    // x_shape = (batch_size, in_channels, d_1, d_2, ..., d_N)
    int64_t batch_size = x.shape()[0];

    // out_shape = (batch_size, out_channels, out_1, out_2, ..., out_N)
    Shape out_shape{batch_size, out_channels};
    for (int8_t i = 0; i < ndim; ++i) {
        out_shape.emplace_back(xchainer::internal::GetConvOutDim(x.shape()[i + 2], w.shape()[i + 2], stride[i], pad[i], cover_all));
        assert(out_shape.back() > 0);
    }
    Array y = Empty(out_shape, x.dtype(), *this);

    cudnn_context_.ConvolutionForward(x, w, b, y, pad, stride, nonstd::nullopt, 1);

    return y;
}

Array CudaDevice::ConvGradWeight(
        Dtype w_dtype,
        const Shape& w_shape,
        const Array& x,
        const Array& gy,
        const StackVector<int64_t, kMaxNdim>& stride,
        const StackVector<int64_t, kMaxNdim>& pad,
        bool cover_all) {
    if (cover_all) {
        throw XchainerError{"CUDA convolution does not support cover_all"};
    }

    CheckDevicesCompatible(x, gy);

    int8_t ndim = x.ndim() - 2;  // Number of spacial dimensions
    if (ndim < 2) {
        throw DimensionError{"CUDA convolution requires number of spacial dimensions to be greater than or equal to 2"};
    }
    assert(x.ndim() == w_shape.ndim());
    assert(stride.size() == static_cast<size_t>(ndim));
    assert(pad.size() == static_cast<size_t>(ndim));
    assert(gy.ndim() == w_shape.ndim());

    Array gw = Empty(w_shape, w_dtype, *this);
    cudnn_context_.ConvolutionBackwardFilter(x, gy, gw, pad, stride, nonstd::nullopt /*dilation*/, 1 /*groups*/);

    return gw;
}

Array CudaDevice::ConvTranspose(
        const Array& x,
        const Array& w,
        const nonstd::optional<Array>& b,
        const StackVector<int64_t, kMaxNdim>& stride,
        const StackVector<int64_t, kMaxNdim>& pad,
        const StackVector<int64_t, kMaxNdim>& out_size) {
    if (b) {
        CheckDevicesCompatible(x, w, *b);
    } else {
        CheckDevicesCompatible(x, w);
    }
    ConvCheckDtype(x, w, b);

    int8_t ndim = x.ndim() - 2;  // Number of spacial dimensions
    if (ndim < 2) {
        throw DimensionError{"CUDA convolution requires number of spacial dimensions to be greater than or equal to 2"};
    }
    assert(w.ndim() == x.ndim());
    assert(stride.size() == static_cast<size_t>(ndim));
    assert(pad.size() == static_cast<size_t>(ndim));
    assert(out_size.size() == static_cast<size_t>(ndim));

    // w.shape = (in_channels, out_channels, k_1, k_2, ..., k_N)
    int64_t out_channels = w.shape()[1];
    // x_shape = (batch_size, in_channels, d_1, d_2, ..., d_N)
    int64_t batch_size = x.shape()[0];

    // out_shape = (batch_size, out_channels, out_1, out_2, ..., out_N)
    // (Note that cover_all is not supported in cuDNN implementation.)
    Shape out_shape{batch_size, out_channels};
    std::copy(out_size.begin(), out_size.end(), std::back_inserter(out_shape));

    Array y = Empty(out_shape, x.dtype(), *this);

    cudnn_context_.ConvolutionBackwardData(w, x, b, y, pad, stride, nonstd::nullopt /*dilation*/, 1 /*group*/);

    return y;
}

void CudaDevice::Synchronize() {
    CheckCudaError(hipSetDevice(index()));
    CheckCudaError(hipDeviceSynchronize());
}

}  // namespace cuda
}  // namespace xchainer
