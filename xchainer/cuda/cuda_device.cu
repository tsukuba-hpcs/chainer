#include "xchainer/cuda/cuda_device.h"

#include <hipblas.h>

#include <algorithm>
#include <cassert>
#include <cmath>
#include <cstddef>
#include <cstdint>
#include <memory>
#include <numeric>

#include <hip/hip_runtime.h>

#include "xchainer/array.h"
#include "xchainer/axes.h"
#include "xchainer/backend_util.h"
#include "xchainer/cuda/cast.cuh"
#include "xchainer/cuda/hipblas.h"
#include "xchainer/cuda/hip/hip_runtime.h"
#include "xchainer/cuda/elementwise.cuh"
#include "xchainer/device.h"
#include "xchainer/dtype.h"
#include "xchainer/enum.h"
#include "xchainer/error.h"
#include "xchainer/indexable_array.h"
#include "xchainer/indexer.h"
#include "xchainer/native/native_device.h"
#include "xchainer/routines/connection.h"
#include "xchainer/routines/creation.h"
#include "xchainer/scalar.h"
#include "xchainer/shape.h"

namespace xchainer {
namespace cuda {

CudaDevice::~CudaDevice() {
    if (cublas_handle_) {
        hipSetDevice(index());
        hipblasDestroy(cublas_handle_);
    }
}

hipblasHandle_t CudaDevice::cublas_handle() {
    if (!cublas_handle_) {
        CheckCudaError(hipSetDevice(index()));
        CheckCublasError(hipblasCreate(&cublas_handle_));
    }
    return cublas_handle_;
}

std::shared_ptr<void> CudaDevice::Allocate(size_t bytesize) {
    CheckCudaError(hipSetDevice(index()));
    void* ptr = memory_pool_.Malloc(bytesize);
    return std::shared_ptr<void>{ptr, [this](void* ptr) { memory_pool_.Free(ptr); }};
}

std::shared_ptr<void> CudaDevice::MakeDataFromForeignPointer(const std::shared_ptr<void>& data) {
    // check memory validity
    void* ptr = data.get();
    hipPointerAttribute_t attr{};
    hipError_t status = hipPointerGetAttributes(&attr, ptr);
    switch (status) {
        case hipSuccess:
            if (attr.isManaged == 0) {
                throw XchainerError{"CUDA memory: ", ptr, " must be a managed (unified) memory"};
            }
            if (attr.device != index()) {
                throw XchainerError{"CUDA memory: ", ptr, " must reside on the device: ", index()};
            }
            break;
        case hipErrorInvalidValue:
            throw XchainerError{"Memory: ", ptr, " is not a CUDA memory"};
        default:
            Throw(status);
    }
    return data;
}

void CudaDevice::MemoryCopyFrom(void* dst, const void* src, size_t bytesize, Device& src_device) {
    assert(bytesize == 0 || IsPointerCudaMemory(dst));
    if (bytesize == 0) {
        return;
    }
    CheckCudaError(hipSetDevice(index()));
    if (&src_device == this || nullptr != dynamic_cast<CudaDevice*>(&src_device)) {
        // Copy between CUDA devices
        CheckCudaError(hipMemcpy(dst, src, bytesize, hipMemcpyDeviceToDevice));
    } else {
        assert(nullptr != dynamic_cast<native::NativeDevice*>(&src_device) &&
               "CudaDevice only supports copy between cuda or native devices.");
        // Copy from native device
        CheckCudaError(hipMemcpy(dst, src, bytesize, hipMemcpyHostToDevice));
    }
}

void CudaDevice::MemoryCopyTo(void* dst, const void* src, size_t bytesize, Device& dst_device) {
    assert(bytesize == 0 || src == nullptr || IsPointerCudaMemory(src));
    if (bytesize == 0) {
        return;
    }
    CheckCudaError(hipSetDevice(index()));
    if (&dst_device == this || nullptr != dynamic_cast<CudaDevice*>(&dst_device)) {
        // Copy between CUDA devices
        CheckCudaError(hipMemcpy(dst, src, bytesize, hipMemcpyDeviceToDevice));
    } else {
        assert(nullptr != dynamic_cast<native::NativeDevice*>(&dst_device) &&
               "CudaDevice only supports copy between cuda or native devices.");
        // Copy to native device
        CheckCudaError(hipMemcpy(dst, src, bytesize, hipMemcpyDeviceToHost));
    }
}

std::shared_ptr<void> CudaDevice::TransferDataFrom(
        Device& src_device, const std::shared_ptr<void>& src_ptr, size_t offset, size_t bytesize) {
    std::shared_ptr<void> dst_ptr = Allocate(bytesize);
    MemoryCopyFrom(dst_ptr.get(), &(static_cast<int8_t*>(src_ptr.get())[offset]), bytesize, src_device);
    return dst_ptr;
}

std::shared_ptr<void> CudaDevice::TransferDataTo(Device& dst_device, const std::shared_ptr<void>& src_ptr, size_t offset, size_t bytesize) {
    std::shared_ptr<void> dst_ptr = dst_device.Allocate(bytesize);
    MemoryCopyTo(dst_ptr.get(), &(static_cast<int8_t*>(src_ptr.get())[offset]), bytesize, dst_device);
    return dst_ptr;
}

std::shared_ptr<void> CudaDevice::FromHostMemory(const std::shared_ptr<void>& src_ptr, size_t bytesize) {
    std::shared_ptr<void> dst_ptr = Allocate(bytesize);
    CheckCudaError(hipMemcpy(dst_ptr.get(), src_ptr.get(), bytesize, hipMemcpyHostToDevice));
    return dst_ptr;
}

void CudaDevice::Synchronize() {
    CheckCudaError(hipSetDevice(index()));
    CheckCudaError(hipDeviceSynchronize());
}

}  // namespace cuda
}  // namespace xchainer
