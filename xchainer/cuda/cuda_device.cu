#include "hip/hip_runtime.h"
#include "xchainer/cuda/cuda_device.h"

#include <algorithm>
#include <cstddef>
#include <cstdint>
#include <memory>
#include <vector>

#include <hip/hip_runtime.h>

#include "xchainer/array.h"
#include "xchainer/cuda/hip/hip_runtime.h"
#include "xchainer/device.h"
#include "xchainer/dtype.h"
#include "xchainer/error.h"
#include "xchainer/indexable_array.h"
#include "xchainer/indexer.h"
#include "xchainer/native/native_device.h"
#include "xchainer/scalar.h"

namespace xchainer {
namespace cuda {

namespace {

static constexpr int kMaxReductionBlockSize = 512;

int64_t RoundUpToPowerOf2(int64_t x) {
    --x;
    x |= x >> 1;
    x |= x >> 2;
    x |= x >> 4;
    x |= x >> 8;
    x |= x >> 16;
    x |= x >> 32;
    return x + 1;
}
}  // namespace

namespace {

template <typename T>
__global__ void FillKernel(IndexableArray<T> out_iarray, T value, Indexer<> indexer) {
    for (int64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < indexer.total_size(); i += blockDim.x * gridDim.x) {
        indexer.Set(i);
        out_iarray[indexer] = value;
    }
}

template <typename T>
__global__ void SumKernel(
        IndexableArray<const T> src_iarray,
        IndexableArray<T> out_iarray,
        Indexer<> src_indexer,
        Indexer<> reduce_indexer,
        Indexer<> out_indexer,
        int reduce_block_size) {
    extern __shared__ __align__(8) uint8_t work_bytes[];
    T* work = reinterpret_cast<T*>(work_bytes);
    int tid = threadIdx.x;
    int reduce_blocks_per_grid = (blockDim.x + reduce_block_size - 1) / reduce_block_size * gridDim.x;

    for (int64_t i_out = blockIdx.x; i_out < out_indexer.total_size(); i_out += gridDim.x * reduce_blocks_per_grid) {
        out_indexer.Set(i_out);

        T sum_value = 0;

        // Set output indices in the corresponding indices (out_axis) in src_index.
        for (int8_t i_out_dim = 0; i_out_dim < out_indexer.ndim(); ++i_out_dim) {
            src_indexer.index()[i_out_dim] = out_indexer.index()[i_out_dim];
        }

        // Linearly compute the partial sum into at most kMaxReductionBlockSize values.
        for (int64_t i_reduce = tid; i_reduce < reduce_indexer.total_size(); i_reduce += reduce_block_size) {
            reduce_indexer.Set(i_reduce);

            // Set reduction indices in the corresponding indices (axis) in src_index.
            for (int8_t i_reduce_dim = 0; i_reduce_dim < reduce_indexer.ndim(); ++i_reduce_dim) {
                src_indexer.index()[out_indexer.ndim() + i_reduce_dim] = reduce_indexer.index()[i_reduce_dim];
            }

            sum_value += src_iarray[src_indexer];
        }

        if (reduce_block_size >= 2) {
            // Synchronize partial sums
            work[tid] = sum_value;
            __syncthreads();

            // Reduction
            if (reduce_block_size > 2) {
                if (reduce_block_size > 4) {
                    if (reduce_block_size > 8) {
                        if (reduce_block_size > 16) {
                            if (reduce_block_size > 32) {
                                if (reduce_block_size > 64) {
                                    if (reduce_block_size > 128) {
                                        if (reduce_block_size > 256) {
                                            static_assert(kMaxReductionBlockSize == 512, "");

                                            if (tid < 256) {
                                                work[tid] += work[tid + 256];
                                            }
                                            __syncthreads();
                                        }
                                        if (tid < 128) {
                                            work[tid] += work[tid + 128];
                                        }
                                        __syncthreads();
                                    }
                                    if (tid < 64) {
                                        work[tid] += work[tid + 64];
                                    }
                                    __syncthreads();
                                }
                                if (tid < 32) {
                                    work[tid] += work[tid + 32];
                                }
                                __syncthreads();
                            }
                            if (tid < 16) {
                                work[tid] += work[tid + 16];
                            }
                            __syncthreads();
                        }
                        if (tid < 8) {
                            work[tid] += work[tid + 8];
                        }
                        __syncthreads();
                    }
                    if (tid < 4) {
                        work[tid] += work[tid + 4];
                    }
                    __syncthreads();
                }
                if (tid < 2) {
                    work[tid] += work[tid + 2];
                }
                __syncthreads();
            }
            sum_value = work[0] + work[1];
        }
        // Store the output value
        if (tid == 0) {
            out_iarray[out_indexer] = sum_value;
        }
    }
}

template <typename T>
__global__ void CopyKernel(IndexableArray<const T> src_iarray, IndexableArray<T> out_iarray, Indexer<> indexer) {
    for (int64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < indexer.total_size(); i += blockDim.x * gridDim.x) {
        indexer.Set(i);
        out_iarray[indexer] = src_iarray[indexer];
    }
}

template <typename T>
__global__ void EqualKernel(
        IndexableArray<const T> lhs_iarray, IndexableArray<const T> rhs_iarray, IndexableArray<bool> out_iarray, Indexer<> indexer) {
    const int64_t total_size = indexer.total_size();
    for (int64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < total_size; i += blockDim.x * gridDim.x) {
        indexer.Set(i);
        out_iarray[indexer] = lhs_iarray[indexer] == rhs_iarray[indexer];
    }
}

template <typename T>
__global__ void AddKernel(
        IndexableArray<const T> lhs_iarray, IndexableArray<const T> rhs_iarray, IndexableArray<T> out_iarray, Indexer<> indexer) {
    const int64_t total_size = indexer.total_size();
    for (int64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < total_size; i += blockDim.x * gridDim.x) {
        indexer.Set(i);
        out_iarray[indexer] = lhs_iarray[indexer] + rhs_iarray[indexer];
    }
}

template <typename T>
__global__ void MulScalarKernel(IndexableArray<const T> lhs_iarray, T rhs_value, IndexableArray<T> out_iarray, Indexer<> indexer) {
    const int64_t total_size = indexer.total_size();
    for (int64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < total_size; i += blockDim.x * gridDim.x) {
        indexer.Set(i);
        out_iarray[indexer] = lhs_iarray[indexer] * rhs_value;
    }
}

template <typename T>
__global__ void MulKernel(
        IndexableArray<const T> lhs_iarray, IndexableArray<const T> rhs_iarray, IndexableArray<T> out_iarray, Indexer<> indexer) {
    const int64_t total_size = indexer.total_size();
    for (int64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < total_size; i += blockDim.x * gridDim.x) {
        indexer.Set(i);
        out_iarray[indexer] = lhs_iarray[indexer] * rhs_iarray[indexer];
    }
}

template <typename T>
__global__ void IfLessElseKernel(
        IndexableArray<const T> lhs_iarray,
        T rhs_value,
        T pos_value,
        IndexableArray<const T> neg_iarray,
        IndexableArray<T> out_iarray,
        Indexer<> indexer) {
    const int64_t total_size = indexer.total_size();
    for (int64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < total_size; i += blockDim.x * gridDim.x) {
        indexer.Set(i);
        out_iarray[indexer] = lhs_iarray[indexer] < rhs_value ? pos_value : neg_iarray[indexer];
    }
}

}  // namespace

std::shared_ptr<void> CudaDevice::Allocate(size_t bytesize) {
    if (bytesize == 0) {
        return nullptr;
    }
    CheckError(hipSetDevice(index()));
    void* raw_ptr = nullptr;
    // Be careful to be exception-safe, i.e.,
    // do not throw any exceptions before creating shared_ptr when memory allocation is succeeded
    hipError_t status = hipMallocManaged(&raw_ptr, bytesize, hipMemAttachGlobal);
    if (status != hipSuccess) {
        cuda::Throw(status);
    }
    return std::shared_ptr<void>{raw_ptr, hipFree};
}

void CudaDevice::MemoryCopyFrom(void* dst, const void* src, size_t bytesize, Device& src_device) {
    assert(IsPointerCudaMemory(dst));
    if (&src_device == this || nullptr != dynamic_cast<CudaDevice*>(&src_device)) {
        // Copy between CUDA devices
        CheckError(hipMemcpy(dst, src, bytesize, hipMemcpyDeviceToDevice));
    } else {
        assert(nullptr != dynamic_cast<native::NativeDevice*>(&src_device) &&
               "CudaDevice only supports copy between cuda or native devices.");
        // Copy from native device
        CheckError(hipMemcpy(dst, src, bytesize, hipMemcpyHostToDevice));
    }
}

void CudaDevice::MemoryCopyTo(void* dst, const void* src, size_t bytesize, Device& dst_device) {
    assert(IsPointerCudaMemory(src));
    if (&dst_device == this || nullptr != dynamic_cast<CudaDevice*>(&dst_device)) {
        // Copy between CUDA devices
        CheckError(hipMemcpy(dst, src, bytesize, hipMemcpyDeviceToDevice));
    } else {
        assert(nullptr != dynamic_cast<native::NativeDevice*>(&dst_device) &&
               "CudaDevice only supports copy between cuda or native devices.");
        // Copy to native device
        CheckError(hipMemcpy(dst, src, bytesize, hipMemcpyDeviceToHost));
    }
}

std::shared_ptr<void> CudaDevice::TransferDataFrom(
        Device& src_device, const std::shared_ptr<void>& src_ptr, size_t offset, size_t bytesize) {
    std::shared_ptr<void> dst_ptr = Allocate(bytesize);
    MemoryCopyFrom(dst_ptr.get(), &(static_cast<int8_t*>(src_ptr.get())[offset]), bytesize, src_device);
    return dst_ptr;
}

std::shared_ptr<void> CudaDevice::TransferDataTo(Device& dst_device, const std::shared_ptr<void>& src_ptr, size_t offset, size_t bytesize) {
    std::shared_ptr<void> dst_ptr = dst_device.Allocate(bytesize);
    MemoryCopyTo(dst_ptr.get(), &(static_cast<int8_t*>(src_ptr.get())[offset]), bytesize, dst_device);
    return dst_ptr;
}

std::shared_ptr<void> CudaDevice::FromContiguousData(const std::shared_ptr<void>& src_ptr, size_t bytesize) {
    std::shared_ptr<void> dst_ptr = Allocate(bytesize);
    CheckError(hipMemcpy(dst_ptr.get(), src_ptr.get(), bytesize, hipMemcpyHostToDevice));
    return dst_ptr;
}

void CudaDevice::Fill(const Array& out, Scalar value) {
    VisitDtype(out.dtype(), [&](auto pt) {
        using T = typename decltype(pt)::type;
        static const int kMaxBlockSize = CudaOccupancyMaxPotentialBlockSize(&FillKernel<T>).block_size;

        IndexableArray<T> out_iarray{out};
        Indexer<> indexer{out.shape()};
        int64_t grid_size = (indexer.total_size() + kMaxBlockSize - 1) / kMaxBlockSize;
        int64_t block_size = std::min<int64_t>(indexer.total_size(), kMaxBlockSize);

        FillKernel<<<grid_size, block_size>>>(out_iarray, static_cast<T>(value), indexer);
    });
}

void CudaDevice::Sum(const Array& src, const std::vector<int8_t>& axis, const Array& out) {
    VisitDtype(out.dtype(), [&](auto pt) {
        using T = typename decltype(pt)::type;
        static const int kMaxBlockSize = CudaOccupancyMaxPotentialBlockSize(&SumKernel<T>).block_size;

        // Prepare indexable arrays and indexers
        auto tup = native::internal::PrepareIndexableArraysForReduction<T>(src, axis, out);
        IndexableArray<const T>& src_iarray = std::get<0>(tup);
        IndexableArray<T>& out_iarray = std::get<1>(tup);
        Indexer<>& src_indexer = std::get<2>(tup);
        Indexer<>& out_indexer = std::get<3>(tup);
        Indexer<>& reduce_indexer = std::get<4>(tup);

        // Launch kernel
        int reduce_block_size =
                static_cast<int>(std::min(static_cast<int64_t>(kMaxReductionBlockSize), RoundUpToPowerOf2(reduce_indexer.total_size())));
        int block_size = std::min(kMaxBlockSize, reduce_block_size);
        int64_t total_reduce_blocks = out_indexer.total_size();
        int64_t grid_size = total_reduce_blocks;
        size_t shared_mem_size = sizeof(T) * reduce_block_size;

        SumKernel<<<grid_size, block_size, shared_mem_size>>>(
                src_iarray, out_iarray, src_indexer, reduce_indexer, out_indexer, reduce_block_size);
    });
}

void CudaDevice::Copy(const Array& src, const Array& out) {
    CheckDevicesCompatible(src, out);
    hipSetDevice(index());
    VisitDtype(out.dtype(), [&](auto pt) {
        using T = typename decltype(pt)::type;
        static const int kMaxBlockSize = CudaOccupancyMaxPotentialBlockSize(&CopyKernel<T>).block_size;

        IndexableArray<const T> src_iarray{src};
        IndexableArray<T> out_iarray{out};
        Indexer<> indexer{out.shape()};

        int64_t total_size = indexer.total_size();
        int64_t grid_size = (total_size + kMaxBlockSize - 1) / kMaxBlockSize;
        int64_t block_size = std::min<int64_t>(total_size, kMaxBlockSize);

        CopyKernel<<<grid_size, block_size>>>(src_iarray, out_iarray, indexer);
    });
}

void CudaDevice::Equal(const Array& lhs, const Array& rhs, const Array& out) {
    CheckDevicesCompatible(lhs, rhs, out);
    hipSetDevice(index());
    VisitDtype(lhs.dtype(), [&](auto pt) {
        using T = typename decltype(pt)::type;
        static const int kMaxBlockSize = CudaOccupancyMaxPotentialBlockSize(&EqualKernel<T>).block_size;

        IndexableArray<const T> lhs_iarray{lhs};
        IndexableArray<const T> rhs_iarray{rhs};
        IndexableArray<bool> out_iarray{out};
        Indexer<> indexer{lhs.shape()};

        int64_t total_size = indexer.total_size();
        int64_t grid_size = (total_size + kMaxBlockSize - 1) / kMaxBlockSize;
        int64_t block_size = std::min<int64_t>(total_size, kMaxBlockSize);

        EqualKernel<<<grid_size, block_size>>>(lhs_iarray, rhs_iarray, out_iarray, indexer);
    });
}

// TODO(sonots): support stream
void CudaDevice::Add(const Array& lhs, const Array& rhs, const Array& out) {
    CheckDevicesCompatible(lhs, rhs, out);
    hipSetDevice(index());
    VisitDtype(lhs.dtype(), [&](auto pt) {
        using T = typename decltype(pt)::type;
        static const int kMaxBlockSize = CudaOccupancyMaxPotentialBlockSize(&AddKernel<T>).block_size;

        IndexableArray<const T> lhs_iarray{lhs};
        IndexableArray<const T> rhs_iarray{rhs};
        IndexableArray<T> out_iarray{out};
        Indexer<> indexer{lhs.shape()};

        int64_t total_size = indexer.total_size();
        int64_t grid_size = (total_size + kMaxBlockSize - 1) / kMaxBlockSize;
        int64_t block_size = std::min<int64_t>(total_size, kMaxBlockSize);

        AddKernel<<<grid_size, block_size>>>(lhs_iarray, rhs_iarray, out_iarray, indexer);
    });
}

void CudaDevice::Mul(const Array& lhs, Scalar rhs, const Array& out) {
    CheckDevicesCompatible(lhs, out);
    hipSetDevice(index());
    VisitDtype(lhs.dtype(), [&](auto pt) {
        using T = typename decltype(pt)::type;
        static const int kMaxBlockSize = CudaOccupancyMaxPotentialBlockSize(&MulScalarKernel<T>).block_size;

        IndexableArray<const T> lhs_iarray{lhs};
        IndexableArray<T> out_iarray{out};
        Indexer<> indexer{lhs.shape()};

        int64_t total_size = indexer.total_size();
        int64_t grid_size = (total_size + kMaxBlockSize - 1) / kMaxBlockSize;
        int64_t block_size = std::min<int64_t>(total_size, kMaxBlockSize);

        MulScalarKernel<<<grid_size, block_size>>>(lhs_iarray, static_cast<T>(rhs), out_iarray, indexer);
    });
}

// TODO(sonots): support stream
void CudaDevice::Mul(const Array& lhs, const Array& rhs, const Array& out) {
    CheckDevicesCompatible(lhs, rhs, out);
    hipSetDevice(index());
    VisitDtype(lhs.dtype(), [&](auto pt) {
        using T = typename decltype(pt)::type;
        static const int kMaxBlockSize = CudaOccupancyMaxPotentialBlockSize(&MulKernel<T>).block_size;

        IndexableArray<const T> lhs_iarray{lhs};
        IndexableArray<const T> rhs_iarray{rhs};
        IndexableArray<T> out_iarray{out};
        Indexer<> indexer{lhs.shape()};

        int64_t total_size = indexer.total_size();
        int64_t grid_size = (total_size + kMaxBlockSize - 1) / kMaxBlockSize;
        int64_t block_size = std::min<int64_t>(total_size, kMaxBlockSize);

        MulKernel<<<grid_size, block_size>>>(lhs_iarray, rhs_iarray, out_iarray, indexer);
    });
}

void CudaDevice::IfLessElse(const Array& lhs, Scalar rhs, Scalar pos, const Array& neg, const Array& out) {
    CheckDevicesCompatible(lhs, neg, out);
    hipSetDevice(index());
    VisitDtype(lhs.dtype(), [&](auto pt) {
        using T = typename decltype(pt)::type;
        static const int kMaxBlockSize = CudaOccupancyMaxPotentialBlockSize(&IfLessElseKernel<T>).block_size;

        IndexableArray<const T> lhs_iarray{lhs};
        IndexableArray<const T> neg_iarray{neg};
        IndexableArray<T> out_iarray{out};
        Indexer<> indexer{lhs.shape()};
        T rhs_value{rhs};
        T pos_value{pos};

        int64_t total_size = indexer.total_size();
        int64_t grid_size = (total_size + kMaxBlockSize - 1) / kMaxBlockSize;
        int64_t block_size = std::min<int64_t>(total_size, kMaxBlockSize);

        IfLessElseKernel<<<grid_size, block_size>>>(lhs_iarray, rhs_value, pos_value, neg_iarray, out_iarray, indexer);
    });
}

void CudaDevice::Dot(const Array& lhs, const Array& rhs, const Array& out) {
    (void)lhs;  // unused
    (void)rhs;  // unused
    (void)out;  // unused
    throw NotImplementedError("CudaDevice::Dot is not yet implemented.");
}

void CudaDevice::Synchronize() {
    CheckError(hipSetDevice(index()));
    CheckError(hipDeviceSynchronize());
}

}  // namespace cuda
}  // namespace xchainer
