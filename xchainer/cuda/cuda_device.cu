#include "hip/hip_runtime.h"
#include "xchainer/cuda/cuda_device.h"

#include <algorithm>
#include <memory>
#include <tuple>
#include <utility>

#include <hip/hip_runtime.h>

#include "xchainer/array.h"
#include "xchainer/cuda/hip/hip_runtime.h"
#include "xchainer/device.h"
#include "xchainer/dtype.h"
#include "xchainer/memory.h"
#include "xchainer/native_backend.h"
#include "xchainer/scalar.h"

namespace xchainer {
namespace cuda {

namespace {

template <typename T>
__global__ void FillKernel(T* odata, T value, int64_t total_size) {
    for (int64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < total_size; i += blockDim.x * gridDim.x) {
        odata[i] = value;
    }
}

template <typename T>
__global__ void AddKernel(const T* ldata, const T* rdata, T* odata, int64_t total_size) {
    for (int64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < total_size; i += blockDim.x * gridDim.x) {
        odata[i] = ldata[i] + rdata[i];
    }
}

template <typename T>
__global__ void MulKernel(const T* ldata, const T* rdata, T* odata, int64_t total_size) {
    for (int64_t i = blockIdx.x * blockDim.x + threadIdx.x; i < total_size; i += blockDim.x * gridDim.x) {
        odata[i] = ldata[i] * rdata[i];
    }
}

}  // namespace

std::shared_ptr<void> CudaDevice::Allocate(size_t bytesize) {
    CheckError(hipSetDevice(index()));
    void* raw_ptr = nullptr;
    // Be careful to be exception-safe, i.e.,
    // do not throw any exceptions before creating shared_ptr when memory allocation is succeeded
    hipError_t status = hipMallocManaged(&raw_ptr, bytesize, hipMemAttachGlobal);
    if (status != hipSuccess) {
        cuda::Throw(status);
    }
    return std::shared_ptr<void>{raw_ptr, hipFree};
}

void CudaDevice::MemoryCopy(void* dst_ptr, const void* src_ptr, size_t bytesize) {
    CheckError(hipSetDevice(index()));
    CheckError(hipMemcpy(dst_ptr, src_ptr, bytesize, hipMemcpyDeviceToDevice));
}

std::shared_ptr<void> CudaDevice::FromBuffer(const std::shared_ptr<void>& src_ptr, size_t bytesize) {
    std::shared_ptr<void> dst_ptr = Allocate(bytesize);
    cuda::CheckError(hipMemcpy(dst_ptr.get(), src_ptr.get(), bytesize, hipMemcpyHostToDevice));
    return dst_ptr;
}

void CudaDevice::Fill(Array& out, Scalar value) {
    VisitDtype(out.dtype(), [&](auto pt) {
        using T = typename decltype(pt)::type;
        static const int kMaxBlockSize = CudaOccupancyMaxPotentialBlockSize(&FillKernel<T>).block_size;

        int64_t total_size = out.GetTotalSize();
        int64_t grid_size = (total_size + kMaxBlockSize - 1) / kMaxBlockSize;
        int64_t block_size = std::min<int64_t>(total_size, kMaxBlockSize);

        auto* odata = static_cast<T*>(out.data().get());
        FillKernel<<<grid_size, block_size>>>(odata, static_cast<T>(value), total_size);
    });
}

// TODO(sonots): support stream
void CudaDevice::Add(const Array& lhs, const Array& rhs, Array& out) {
    CheckDevicesCompatible(lhs, rhs, out);
    hipSetDevice(index());
    VisitDtype(lhs.dtype(), [&](auto pt) {
        using T = typename decltype(pt)::type;
        static const int kMaxBlockSize = CudaOccupancyMaxPotentialBlockSize(&AddKernel<T>).block_size;

        int64_t total_size = lhs.GetTotalSize();
        int64_t grid_size = (total_size + kMaxBlockSize - 1) / kMaxBlockSize;
        int64_t block_size = std::min<int64_t>(total_size, kMaxBlockSize);

        auto* ldata = static_cast<const T*>(lhs.data().get());
        auto* rdata = static_cast<const T*>(rhs.data().get());
        auto* odata = static_cast<T*>(out.data().get());
        AddKernel<<<grid_size, block_size>>>(ldata, rdata, odata, total_size);
    });
}

// TODO(sonots): support stream
void CudaDevice::Mul(const Array& lhs, const Array& rhs, Array& out) {
    CheckDevicesCompatible(lhs, rhs, out);
    hipSetDevice(index());
    VisitDtype(lhs.dtype(), [&](auto pt) {
        using T = typename decltype(pt)::type;
        static const int kMaxBlockSize = CudaOccupancyMaxPotentialBlockSize(&MulKernel<T>).block_size;

        int64_t total_size = lhs.GetTotalSize();
        int64_t grid_size = (total_size + kMaxBlockSize - 1) / kMaxBlockSize;
        int64_t block_size = std::min<int64_t>(total_size, kMaxBlockSize);

        auto* ldata = static_cast<const T*>(lhs.data().get());
        auto* rdata = static_cast<const T*>(rhs.data().get());
        auto* odata = static_cast<T*>(out.data().get());
        MulKernel<<<grid_size, block_size>>>(ldata, rdata, odata, total_size);
    });
}

void CudaDevice::Synchronize() {
    CheckError(hipSetDevice(index()));
    CheckError(hipDeviceSynchronize());
}

std::tuple<std::shared_ptr<void>, size_t> CudaDevice::TransferDataFrom(Device& src_device, const std::shared_ptr<void>& src_ptr,
                                                                       size_t offset, size_t bytesize) {
    (void)src_device;  // unused

    // src_device is either a native device or a CUDA device, so the direct access is always possible.
    std::shared_ptr<void> dst_ptr = Allocate(bytesize);
    MemoryCopy(dst_ptr.get(), &static_cast<int8_t*>(src_ptr.get())[offset], bytesize);
    return std::make_tuple(std::move(dst_ptr), 0);
}

std::tuple<std::shared_ptr<void>, size_t> CudaDevice::TransferDataTo(Device& dst_device, const std::shared_ptr<void>& src_ptr,
                                                                     size_t offset, size_t bytesize) {
    if (dst_device.backend().GetName() == NativeBackend::kDefaultName) {
        // Synchronize the source data on CUDA device so that the native device can read it.
        Synchronize();
    }

    std::shared_ptr<void> dst_ptr = dst_device.Allocate(bytesize);
    dst_device.MemoryCopy(dst_ptr.get(), &static_cast<int8_t*>(src_ptr.get())[offset], bytesize);
    return std::make_tuple(std::move(dst_ptr), 0);
}

}  // namespace cuda
}  // namespace xchainer
