#include "xchainer/cuda/cuda_device.h"

#include <hipblas.h>

#include <algorithm>
#include <cassert>
#include <cmath>
#include <cstddef>
#include <cstdint>
#include <memory>
#include <numeric>

#include <hip/hip_runtime.h>

#include "xchainer/array.h"
#include "xchainer/axes.h"
#include "xchainer/backend_util.h"
#include "xchainer/cuda/cast.cuh"
#include "xchainer/cuda/hipblas.h"
#include "xchainer/cuda/hip/hip_runtime.h"
#include "xchainer/cuda/elementwise.cuh"
#include "xchainer/device.h"
#include "xchainer/dtype.h"
#include "xchainer/enum.h"
#include "xchainer/error.h"
#include "xchainer/indexable_array.h"
#include "xchainer/indexer.h"
#include "xchainer/native/native_device.h"
#include "xchainer/routines/connection.h"
#include "xchainer/routines/creation.h"
#include "xchainer/scalar.h"
#include "xchainer/shape.h"

namespace xchainer {
namespace cuda {

CudaDevice::~CudaDevice() {
    if (cublas_handle_) {
        hipSetDevice(index());
        hipblasDestroy(cublas_handle_);
    }
}

hipblasHandle_t CudaDevice::cublas_handle() {
    if (!cublas_handle_) {
        CheckCudaError(hipSetDevice(index()));
        CheckCublasError(hipblasCreate(&cublas_handle_));
    }
    return cublas_handle_;
}

void CudaDevice::Synchronize() {
    CheckCudaError(hipSetDevice(index()));
    CheckCudaError(hipDeviceSynchronize());
}

}  // namespace cuda
}  // namespace xchainer
