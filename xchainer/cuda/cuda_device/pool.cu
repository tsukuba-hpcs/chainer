#include "hip/hip_runtime.h"
#include "xchainer/cuda/cuda_device.h"

#include <cassert>
#include <cstdint>
#include <memory>
#include <utility>

#include <hipDNN.h>

#include "xchainer/array.h"
#include "xchainer/backend_util.h"
#include "xchainer/constant.h"
#include "xchainer/cuda/hip/hip_runtime.h"
#include "xchainer/cuda/hipDNN.h"
#include "xchainer/device.h"
#include "xchainer/dtype.h"
#include "xchainer/error.h"
#include "xchainer/indexable_array.h"
#include "xchainer/indexer.h"
#include "xchainer/numeric_limits.h"
#include "xchainer/routines/connection.h"
#include "xchainer/routines/creation.h"
#include "xchainer/routines/pooling.h"
#include "xchainer/shape.h"
#include "xchainer/stack_vector.h"

namespace xchainer {
namespace cuda {
namespace {

// Uses the previously computed y to find the indices for which the upstream gradients should be propagated.
// It is faster than looking for the argmax again since we only have to do a single comparison.
// TODO(hvy): Make the spatial dimensionality a template parameter to allow unrolling the loops.
template <typename T>
__global__ void MaxPoolDoubleBackwardKernel(
        IndexableArray<const T> ggx_iarray,
        IndexableArray<const T> x_iarray,
        IndexableArray<const T> y_iarray,
        IndexableArray<T> ggy_iarray,
        Indexer<> x_indexer,
        Indexer<> y_indexer,
        Indexer<> kernel_indexer,
        int64_t* stride,
        int64_t* pad,
        NdimIndex x_index) {
    for (auto it_y = y_indexer.It(blockIdx.x * blockDim.x + threadIdx.x, blockDim.x * gridDim.x); it_y; ++it_y) {
        x_index.index()[0] = it_y.index()[0];  // batch.
        x_index.index()[1] = it_y.index()[1];  // channel.

        T y = y_iarray[it_y];

        // Iterate over the kernel in the reverse order, since the resulting index should the be first match.
        for (auto it_kernel = kernel_indexer.It(kernel_indexer.total_size() - 1); it_kernel.raw_index() >= 0; --it_kernel) {
            for (int8_t i = 2; i < x_indexer.ndim(); ++i) {
                int64_t idx = it_y.index()[i] * stride[i - 2] - pad[i - 2] + it_kernel.index()[i - 2];
                idx = max(idx, int64_t{0});
                idx = min(idx, x_indexer.shape()[i] - 1);
                x_index.index()[i] = idx;
            }
            auto it_x = x_indexer.At(x_index);
            if (y == x_iarray[it_x]) {
                ggy_iarray[it_y] = ggx_iarray[it_x];
            }
        }
    }
}

class PoolImpl {
public:
    PoolImpl(
            hipdnnHandle_t cudnn_handle,
            StackVector<int64_t, kMaxNdim> kernel_size,
            StackVector<int64_t, kMaxNdim> stride,
            StackVector<int64_t, kMaxNdim> pad,
            bool cover_all,
            hipdnnPoolingMode_t cudnn_pooling_mode)
        : cudnn_handle_{cudnn_handle},
          kernel_size_{std::move(kernel_size)},
          stride_{std::move(stride)},
          pad_{std::move(pad)},
          cover_all_{cover_all},
          cudnn_pooling_mode_{cudnn_pooling_mode} {
        if (cover_all_) {
            throw XchainerError{"CUDA pooling does not support cover_all"};
        }
    }

    Array Forward(const Array& x) {
        int8_t ndim = x.ndim() - 2;  // Number of spacial dimensions
        if (ndim != 2 && ndim != 3) {
            throw DimensionError{"XChainer cuDNN pooling supports only 2 and 3 spatial dimensions."};
        }

        assert(kernel_size_.size() == static_cast<size_t>(ndim));
        assert(stride_.size() == static_cast<size_t>(ndim));
        assert(pad_.size() == static_cast<size_t>(ndim));

        // out_shape = (batch_size, out_channels, out_1, out_2, ..., out_N)
        Shape out_shape{x.shape()[0], x.shape()[1]};
        for (int8_t i = 0; i < ndim; ++i) {
            out_shape.emplace_back(xchainer::internal::GetConvOutDim(x.shape()[i + 2], kernel_size_[i], stride_[i], pad_[i], cover_all_));
            assert(out_shape.back() > 0);
        }

        Array y = Empty(out_shape, x.dtype(), x.device());
        Array x_cont = AsContiguousArray(x);

        internal::CudnnTensorDescriptor x_desc{x_cont};
        internal::CudnnTensorDescriptor y_desc{y};

        internal::CudnnPoolingDescriptor pool_desc{cudnn_pooling_mode_, HIPDNN_NOT_PROPAGATE_NAN, kernel_size_, pad_, stride_};

        CheckCudnnError(hipdnnPoolingForward(
                cudnn_handle_,
                *pool_desc,
                internal::GetValuePtr<1>(x.dtype()),
                *x_desc,
                xchainer::internal::GetRawOffsetData<void>(x_cont),
                internal::GetValuePtr<0>(x.dtype()),
                *y_desc,
                xchainer::internal::GetRawOffsetData<void>(y)));

        x_ = x.AsConstant();
        y_ = y.AsConstant();

        return y;
    }

    Array Backward(const Array& gout) {
        int8_t ndim = x_.ndim() - 2;  // Number of spacial dimensions
        if (ndim < 2) {
            throw DimensionError{"CUDA pooling requires number of spatial dimensions to be greater than or equal to 2"};
        }

        assert(kernel_size_.size() == static_cast<size_t>(ndim));
        assert(stride_.size() == static_cast<size_t>(ndim));
        assert(pad_.size() == static_cast<size_t>(ndim));
        assert(gout.shape() == y_.shape());

        Array gx = EmptyLike(x_, x_.device());
        Array y_cont = AsContiguousArray(y_);
        Array gout_cont = AsContiguousArray(gout);
        Array x_cont = AsContiguousArray(x_);

        internal::CudnnTensorDescriptor y_desc{y_cont};
        internal::CudnnTensorDescriptor gout_desc{gout_cont};
        internal::CudnnTensorDescriptor x_desc{x_cont};
        internal::CudnnTensorDescriptor gx_desc{gx};

        internal::CudnnPoolingDescriptor pool_desc{cudnn_pooling_mode_, HIPDNN_NOT_PROPAGATE_NAN, kernel_size_, pad_, stride_};

        CheckCudnnError(hipdnnPoolingBackward(
                cudnn_handle_,
                *pool_desc,
                internal::GetValuePtr<1>(x_.dtype()),
                *y_desc,
                xchainer::internal::GetRawOffsetData<void>(y_cont),
                *gout_desc,
                xchainer::internal::GetRawOffsetData<void>(gout_cont),
                *x_desc,
                xchainer::internal::GetRawOffsetData<void>(x_cont),
                internal::GetValuePtr<0>(x_.dtype()),
                *gx_desc,
                xchainer::internal::GetRawOffsetData<void>(gx)));

        return gx;
    }

    Array DoubleBackward(const Array& ggx) {
        Device& device = ggx.device();
        Array ggy = EmptyLike(y_, y_.device());

        VisitDtype(ggy.dtype(), [&](auto pt) {
            using T = typename decltype(pt)::type;

            IndexableArray<const T> ggx_iarray{ggx};
            IndexableArray<const T> x_iarray{x_};
            IndexableArray<const T> y_iarray{y_};
            IndexableArray<T> ggy_iarray{ggy};

            Indexer<> x_indexer{x_.shape()};
            Indexer<> y_indexer{y_.shape()};
            Indexer<> kernel_indexer{Shape{kernel_size_.begin(), kernel_size_.end()}};

            std::shared_ptr<void> stride = device.Allocate(stride_.size() * sizeof(int64_t));
            CheckCudaError(hipMemcpy(stride.get(), stride_.data(), stride_.size() * sizeof(int64_t), hipMemcpyHostToDevice));

            std::shared_ptr<void> pad = device.Allocate(pad_.size() * sizeof(int64_t));
            CheckCudaError(hipMemcpy(pad.get(), pad_.data(), pad_.size() * sizeof(int64_t), hipMemcpyHostToDevice));

            static const int kMaxBlockSize = CudaOccupancyMaxPotentialBlockSize(&MaxPoolDoubleBackwardKernel<T>).block_size;
            int64_t total_size = y_indexer.total_size();
            int64_t grid_size = (total_size + kMaxBlockSize - 1) / kMaxBlockSize;
            int64_t block_size = std::min<int64_t>(total_size, kMaxBlockSize);

            MaxPoolDoubleBackwardKernel<<<grid_size, block_size>>>(
                    ggx_iarray,
                    x_iarray,
                    y_iarray,
                    ggy_iarray,
                    x_indexer,
                    y_indexer,
                    kernel_indexer,
                    static_cast<int64_t*>(stride.get()),
                    static_cast<int64_t*>(pad.get()),
                    NdimIndex{x_iarray.ndim()});
        });

        return ggy;
    }

private:
    hipdnnHandle_t cudnn_handle_;
    const StackVector<int64_t, kMaxNdim> kernel_size_;
    const StackVector<int64_t, kMaxNdim> stride_;
    const StackVector<int64_t, kMaxNdim> pad_;
    bool cover_all_;
    hipdnnPoolingMode_t cudnn_pooling_mode_;
    Array x_;
    Array y_;
};

class CudaMaxPoolForwardBackward : public xchainer::MaxPoolForwardBackward {
public:
    explicit CudaMaxPoolForwardBackward(
            hipdnnHandle_t cudnn_handle,
            const StackVector<int64_t, kMaxNdim>& kernel_size,
            const StackVector<int64_t, kMaxNdim>& stride,
            const StackVector<int64_t, kMaxNdim>& pad,
            bool cover_all)
        : pool_impl_{cudnn_handle, kernel_size, stride, pad, cover_all, HIPDNN_POOLING_MAX} {}

    Array Forward(const Array& x) override { return pool_impl_.Forward(x); }

    Array Backward(const Array& gout) override { return pool_impl_.Backward(gout); }

    Array DoubleBackward(const Array& ggx) override { return pool_impl_.DoubleBackward(ggx); }

private:
    PoolImpl pool_impl_;
};

}  // namespace

std::unique_ptr<MaxPoolForwardBackward> CudaDevice::GetMaxPoolForwardBackward(
        const StackVector<int64_t, kMaxNdim>& kernel_size,
        const StackVector<int64_t, kMaxNdim>& stride,
        const StackVector<int64_t, kMaxNdim>& pad,
        bool cover_all) {
    return std::make_unique<CudaMaxPoolForwardBackward>(cudnn_handle(), kernel_size, stride, pad, cover_all);
}

namespace {

hipdnnPoolingMode_t GetCudnnPoolingMode(AveragePoolPadMode pad_mode) {
    switch (pad_mode) {
        case AveragePoolPadMode::kZero:
            return HIPDNN_POOLING_AVERAGE_COUNT_INCLUDE_PADDING;
        case AveragePoolPadMode::kIgnore:
            return HIPDNN_POOLING_AVERAGE_COUNT_EXCLUDE_PADDING;
        default:
            XCHAINER_NEVER_REACH();
    }
}

class CudaAveragePoolForwardBackward : public xchainer::AveragePoolForwardBackward {
public:
    explicit CudaAveragePoolForwardBackward(
            hipdnnHandle_t cudnn_handle,
            const StackVector<int64_t, kMaxNdim>& kernel_size,
            const StackVector<int64_t, kMaxNdim>& stride,
            const StackVector<int64_t, kMaxNdim>& pad,
            AveragePoolPadMode pad_mode)
        : pool_impl_{cudnn_handle, kernel_size, stride, pad, false, GetCudnnPoolingMode(pad_mode)} {}

    Array Forward(const Array& x) override { return pool_impl_.Forward(x); }

    Array Backward(const Array& gout) override { return pool_impl_.Backward(gout); }

private:
    PoolImpl pool_impl_;
};

}  // namespace

std::unique_ptr<AveragePoolForwardBackward> CudaDevice::GetAveragePoolForwardBackward(
        const StackVector<int64_t, kMaxNdim>& kernel_size,
        const StackVector<int64_t, kMaxNdim>& stride,
        const StackVector<int64_t, kMaxNdim>& pad,
        AveragePoolPadMode pad_mode) {
    return std::make_unique<CudaAveragePoolForwardBackward>(cudnn_handle(), kernel_size, stride, pad, pad_mode);
}

}  // namespace cuda
}  // namespace xchainer
