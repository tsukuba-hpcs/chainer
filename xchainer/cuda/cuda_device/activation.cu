#include "xchainer/cuda/cuda_device.h"

#include <cstdint>

#include <hip/hip_runtime.h>

#include "xchainer/array.h"
#include "xchainer/cuda/hip/hip_runtime.h"
#include "xchainer/cuda/elementwise.cuh"
#include "xchainer/device.h"
#include "xchainer/dtype.h"
#include "xchainer/scalar.h"

namespace xchainer {
namespace cuda {

namespace {

template <typename T>
struct IfLessElseASSAImpl {
    __device__ void operator()(int64_t /*i*/, T x1, T neg, T& out) { out = x1 < x2 ? pos : neg; }
    T x2;
    T pos;
};

}  // namespace

void CudaDevice::IfLessElseASSA(const Array& x1, Scalar x2, Scalar pos, const Array& neg, const Array& out) {
    CheckDevicesCompatible(x1, neg, out);
    CheckCudaError(hipSetDevice(index()));
    VisitDtype(out.dtype(), [&](auto pt) {
        using T = typename decltype(pt)::type;
        Elementwise<const T, const T, T>(IfLessElseASSAImpl<T>{static_cast<T>(x2), static_cast<T>(pos)}, x1, neg, out);
    });
}

}  // namespace cuda
}  // namespace xchainer
