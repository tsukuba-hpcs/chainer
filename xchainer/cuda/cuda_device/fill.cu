#include "hip/hip_runtime.h"
#include "xchainer/cuda/cuda_device.h"

#include <algorithm>
#include <cstdint>
#include <mutex>

#include <hip/hip_runtime.h>

#include "xchainer/array.h"
#include "xchainer/cuda/cast.cuh"
#include "xchainer/cuda/hip/hip_runtime.h"
#include "xchainer/cuda/hip/hip_runtime.h"
#include "xchainer/cuda/elementwise.cuh"
#include "xchainer/dtype.h"
#include "xchainer/indexable_array.h"
#include "xchainer/indexer.h"
#include "xchainer/macro.h"
#include "xchainer/scalar.h"
#include "xchainer/shape.h"

namespace xchainer {
namespace cuda {
namespace {

template <typename T>
struct ArangeImpl {
    __device__ void operator()(int64_t i, T& out) { out = start + step * i; }
    T start;
    T step;
};

}  // namespace

void CudaDevice::Arange(Scalar start, Scalar step, const Array& out) {
    CheckCudaError(hipSetDevice(index()));
    VisitDtype(out.dtype(), [&](auto pt) {
        using T = typename decltype(pt)::type;
        Elementwise<T>(ArangeImpl<T>{static_cast<T>(start), static_cast<T>(step)}, out);
    });
}

namespace {

template <typename T>
struct FillImpl {
    __device__ void operator()(int64_t /*i*/, T& out) { out = value; }
    T value;
};

}  // namespace

void CudaDevice::Fill(const Array& out, Scalar value) {
    CheckCudaError(hipSetDevice(index()));
    VisitDtype(out.dtype(), [&](auto pt) {
        using T = typename decltype(pt)::type;
        Elementwise<T>(FillImpl<T>{static_cast<T>(value)}, out);
    });
}

namespace {

template <typename T>
struct IdentityImpl {
    explicit IdentityImpl(int64_t n) : n_plus_one{n + 1} {}
    __device__ void operator()(int64_t i, T& out) { out = i % n_plus_one == 0 ? T{1} : T{0}; }
    int64_t n_plus_one;
};

}  // namespace

void CudaDevice::Identity(const Array& out) {
    XCHAINER_ASSERT(out.ndim() == 2);
    XCHAINER_ASSERT(out.shape()[0] == out.shape()[1]);

    CheckCudaError(hipSetDevice(index()));
    VisitDtype(out.dtype(), [&](auto pt) {
        using T = typename decltype(pt)::type;
        Elementwise<T>(IdentityImpl<T>{out.shape()[0]}, out);
    });
}

namespace {

template <typename T>
struct EyeImpl {
    EyeImpl(int64_t m, int64_t k) : start{k < 0 ? -k * m : k}, stop{m * (m - k)}, step{m + 1} {}
    __device__ void operator()(int64_t i, T& out) { out = start <= i && i < stop && (i - start) % step == 0 ? T{1} : T{0}; }
    int64_t start;
    int64_t stop;
    int64_t step;
};

}  // namespace

void CudaDevice::Eye(int64_t k, const Array& out) {
    CheckCudaError(hipSetDevice(index()));
    VisitDtype(out.dtype(), [k, &out](auto pt) {
        using T = typename decltype(pt)::type;
        Elementwise<T>(EyeImpl<T>{out.shape()[1], k}, out);
    });
}

namespace {

template <typename T>
__global__ void SetVecInMat(
        IndexableArray<const T, 1> vec_iarray,
        IndexableArray<T, 2> mat_iarray,
        Indexer<1> vec_indexer,
        Indexer<1> mat_row_indexer,
        Indexer<1> mat_col_indexer,
        Indexer<2> mat_indexer,
        int64_t mat_row_start,
        int64_t mat_col_start) {
    for (auto vec_it = vec_indexer.It(blockIdx.x * blockDim.x + threadIdx.x, blockDim.x * gridDim.x); vec_it; ++vec_it) {
        auto mat_row_it = mat_row_indexer.It(mat_row_start + vec_it.raw_index());
        auto mat_col_it = mat_col_indexer.It(mat_col_start + vec_it.raw_index());
        auto mat_it = mat_indexer.At(mat_row_it, mat_col_it);
        mat_iarray[mat_it] = vec_iarray[vec_it];
    }
}

}  // namespace

void CudaDevice::Diagflat(const Array& v, int64_t k, const Array& out) {
    XCHAINER_ASSERT(v.ndim() == 1);
    XCHAINER_ASSERT(out.ndim() == 2);

    CheckCudaError(hipSetDevice(index()));
    VisitDtype(out.dtype(), [&](auto pt) {
        using T = typename decltype(pt)::type;

        // Start indices for the 2-D array axes with applied offset k.
        int64_t row_start{0};
        int64_t col_start{0};

        if (k >= 0) {
            col_start += k;
        } else {
            row_start -= k;
        }

        // Initialize all elements to 0 first instead of conditionally filling in the diagonal.
        Fill(out, T{0});

        IndexableArray<const T, 1> v_iarray{v};
        IndexableArray<T, 2> out_iarray{out};
        Indexer<1> v_indexer{v.shape()};
        Indexer<1> out_row_indexer{Shape{out.shape()[0]}};
        Indexer<1> out_col_indexer{Shape{out.shape()[1]}};
        Indexer<2> out_indexer{out.shape()};

        // TODO(niboshi): Calculate kMaxBlockSize per device
        std::lock_guard<std::mutex> lock{*cuda_internal::g_mutex};
        static const int kMaxBlockSize = CudaOccupancyMaxPotentialBlockSize(&SetVecInMat<T>).block_size;
        int64_t total_size = out_indexer.total_size();
        int64_t grid_size = (total_size + kMaxBlockSize - 1) / kMaxBlockSize;
        int64_t block_size = std::min<int64_t>(total_size, kMaxBlockSize);

        SetVecInMat<<<grid_size, block_size>>>(
                v_iarray, out_iarray, v_indexer, out_row_indexer, out_col_indexer, out_indexer, row_start, col_start);
    });
}

namespace {

template <typename T>
struct LinspaceImpl {
    __device__ void operator()(int64_t i, T& out) {
        double value = n == 1 ? start : (start * (n - 1 - i) + stop * i) / (n - 1);
        out = cuda_numeric_cast<T>(value);
    }
    int64_t n;
    double start;
    double stop;
};

}  // namespace

void CudaDevice::Linspace(double start, double stop, const Array& out) {
    XCHAINER_ASSERT(out.ndim() == 1);
    XCHAINER_ASSERT(out.shape()[0] > 0);

    CheckCudaError(hipSetDevice(index()));
    VisitDtype(out.dtype(), [&](auto pt) {
        using T = typename decltype(pt)::type;
        int64_t n = out.shape()[0];
        Elementwise<T>(LinspaceImpl<T>{n, start, stop}, out);
    });
}

}  // namespace cuda
}  // namespace xchainer
