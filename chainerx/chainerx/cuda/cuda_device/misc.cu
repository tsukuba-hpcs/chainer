#include "hip/hip_runtime.h"
#include "chainerx/cuda/cuda_device.h"

#include <cmath>
#include <cstdint>

#include <hip/hip_runtime.h>

#include "chainerx/array.h"
#include "chainerx/cuda/hip/hip_runtime.h"
#include "chainerx/cuda/elementwise.cuh"
#include "chainerx/device.h"
#include "chainerx/dtype.h"

namespace chainerx {
namespace cuda {
namespace {

template <typename T>
struct SqrtImpl {
    __device__ void operator()(int64_t /*i*/, T x, T& out) { out = std::sqrt(x); }
};

}  // namespace

void CudaDevice::Sqrt(const Array& x, const Array& out) {
    CheckDevicesCompatible(x, out);
    CheckCudaError(hipSetDevice(index()));
    VisitFloatingPointDtype(out.dtype(), [&](auto pt) {
        using T = typename decltype(pt)::type;
        Elementwise<const T, T>(SqrtImpl<T>{}, x, out);
    });
}

}  // namespace cuda
}  // namespace chainerx
