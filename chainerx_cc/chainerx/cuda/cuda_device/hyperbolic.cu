#include "chainerx/cuda/cuda_device.h"

#include <cmath>
#include <cstdint>

#include <hip/hip_runtime.h>

#include "chainerx/array.h"
#include "chainerx/cuda/hip/hip_runtime.h"
#include "chainerx/cuda/cuda_set_device_scope.h"
#include "chainerx/cuda/elementwise.cuh"
#include "chainerx/cuda/numeric.cuh"
#include "chainerx/cuda/op_regist.h"
#include "chainerx/device.h"
#include "chainerx/dtype.h"
#include "chainerx/numeric.h"
#include "chainerx/routines/math.h"
#include "chainerx/scalar.h"

namespace chainerx {
namespace cuda {
namespace {

template <typename T>
struct CudaUnaryOp {
    T (*func)(T);

    CudaUnaryOp(T (*func)(T)) : func{func} {}

    __device__ void operator()(int64_t /*i*/, CudaType x, CudaType& out) { out = func(x); }
};

class CudaSinhOp : public SinhOp {
public:
    void Call(const Array& x, const Array& out) override {
        Device& device = x.device();
        device.CheckDevicesCompatible(x, out);
        CudaSetDeviceScope scope{device.index()};
        const Array& x_cast = x.dtype() == out.dtype() ? x : x.AsType(out.dtype());
        VisitFloatingPointDtype(out.dtype(), [&](auto pt) {
            using T = typename decltype(pt)::type;
            using CudaType = cuda_internal::DataType<T>;
            Elementwise<const T, T>(CudaUnaryOp<CudaType>{cuda::Sinh}, x_cast, out);
        });
    }
};

CHAINERX_REGISTER_OP_CUDA(SinhOp, CudaSinhOp);

class CudaCoshOp : public CoshOp {
public:
    void Call(const Array& x, const Array& out) override {
        Device& device = x.device();
        device.CheckDevicesCompatible(x, out);
        CudaSetDeviceScope scope{device.index()};
        const Array& x_cast = x.dtype() == out.dtype() ? x : x.AsType(out.dtype());
        VisitFloatingPointDtype(out.dtype(), [&](auto pt) {
            using T = typename decltype(pt)::type;
            using CudaType = cuda_internal::DataType<T>;
            Elementwise<const T, T>(CudaUnaryOp<CudaType>{cuda::Cosh}, x_cast, out);
        });
    }
};

CHAINERX_REGISTER_OP_CUDA(CoshOp, CudaCoshOp);

class CudaArcsinhOp : public ArcsinhOp {
public:
    void Call(const Array& x, const Array& out) override {
        Device& device = x.device();
        device.CheckDevicesCompatible(x, out);
        CudaSetDeviceScope scope{device.index()};
        const Array& x_cast = x.dtype() == out.dtype() ? x : x.AsType(out.dtype());
        VisitFloatingPointDtype(out.dtype(), [&](auto pt) {
            using T = typename decltype(pt)::type;
            using CudaType = cuda_internal::DataType<T>;
            Elementwise<const T, T>(CudaUnaryOp<CudaType>{cuda::Arcsinh}, x_cast, out);
        });
    }
};

CHAINERX_REGISTER_OP_CUDA(ArcsinhOp, CudaArcsinhOp);

class CudaArccoshOp : public ArccoshOp {
public:
    void Call(const Array& x, const Array& out) override {
        Device& device = x.device();
        device.CheckDevicesCompatible(x, out);
        CudaSetDeviceScope scope{device.index()};
        const Array& x_cast = x.dtype() == out.dtype() ? x : x.AsType(out.dtype());
        VisitFloatingPointDtype(out.dtype(), [&](auto pt) {
            using T = typename decltype(pt)::type;
            using CudaType = cuda_internal::DataType<T>;
            Elementwise<const T, T>(CudaUnaryOp<CudaType>{cuda::Arccosh}, x_cast, out);
        });
    }
};

CHAINERX_REGISTER_OP_CUDA(ArccoshOp, CudaArccoshOp);

}  // namespace
}  // namespace cuda
}  // namespace chainerx
