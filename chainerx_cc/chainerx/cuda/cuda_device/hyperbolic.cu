#include "chainerx/cuda/cuda_device.h"

#include <cmath>
#include <cstdint>

#include <hip/hip_runtime.h>

#include "chainerx/array.h"
#include "chainerx/cuda/hip/hip_runtime.h"
#include "chainerx/cuda/cuda_set_device_scope.h"
#include "chainerx/cuda/elementwise.cuh"
#include "chainerx/cuda/numeric.cuh"
#include "chainerx/cuda/op_regist.h"
#include "chainerx/device.h"
#include "chainerx/dtype.h"
#include "chainerx/numeric.h"
#include "chainerx/routines/math.h"
#include "chainerx/scalar.h"

namespace chainerx {
namespace cuda {
namespace {

template <typename CudaType, typename Op>
struct UnaryOpImpl {
    Op op;

    explicit UnaryOpImpl(Op op) : op{op} {}

    __device__ inline void operator()(int64_t /*i*/, CudaType x, CudaType& out) { out = op(x); }
};

class CudaSinhOp : public SinhOp {
public:
    void Call(const Array& x, const Array& out) override {
        Device& device = x.device();
        device.CheckDevicesCompatible(x, out);
        CudaSetDeviceScope scope{device.index()};
        const Array& x_cast = x.dtype() == out.dtype() ? x : x.AsType(out.dtype());
        VisitFloatingPointDtype(out.dtype(), [&](auto pt) {
            using T = typename decltype(pt)::type;
            using CudaType = cuda_internal::DataType<T>;
            auto op = cuda::Sinh<CudaType>{};
            auto functor = UnaryOpImpl<CudaType, decltype(op)>{op};
            Elementwise<const T, T>(functor, x_cast, out);
        });
    }
};

CHAINERX_REGISTER_OP_CUDA(SinhOp, CudaSinhOp);

class CudaCoshOp : public CoshOp {
public:
    void Call(const Array& x, const Array& out) override {
        Device& device = x.device();
        device.CheckDevicesCompatible(x, out);
        CudaSetDeviceScope scope{device.index()};
        const Array& x_cast = x.dtype() == out.dtype() ? x : x.AsType(out.dtype());
        VisitFloatingPointDtype(out.dtype(), [&](auto pt) {
            using T = typename decltype(pt)::type;
            using CudaType = cuda_internal::DataType<T>;
            auto op = cuda::Cosh<CudaType>{};
            auto functor = UnaryOpImpl<CudaType, decltype(op)>{op};
            Elementwise<const T, T>(functor, x_cast, out);
        });
    }
};

CHAINERX_REGISTER_OP_CUDA(CoshOp, CudaCoshOp);

class CudaArcsinhOp : public ArcsinhOp {
public:
    void Call(const Array& x, const Array& out) override {
        Device& device = x.device();
        device.CheckDevicesCompatible(x, out);
        CudaSetDeviceScope scope{device.index()};
        const Array& x_cast = x.dtype() == out.dtype() ? x : x.AsType(out.dtype());
        VisitFloatingPointDtype(out.dtype(), [&](auto pt) {
            using T = typename decltype(pt)::type;
            using CudaType = cuda_internal::DataType<T>;
            auto op = cuda::Arcsinh<CudaType>{};
            auto functor = UnaryOpImpl<CudaType, decltype(op)>{op};
            Elementwise<const T, T>(functor, x_cast, out);
        });
    }
};

CHAINERX_REGISTER_OP_CUDA(ArcsinhOp, CudaArcsinhOp);

class CudaArccoshOp : public ArccoshOp {
public:
    void Call(const Array& x, const Array& out) override {
        Device& device = x.device();
        device.CheckDevicesCompatible(x, out);
        CudaSetDeviceScope scope{device.index()};
        const Array& x_cast = x.dtype() == out.dtype() ? x : x.AsType(out.dtype());
        VisitFloatingPointDtype(out.dtype(), [&](auto pt) {
            using T = typename decltype(pt)::type;
            using CudaType = cuda_internal::DataType<T>;
            auto op = cuda::Arccosh<CudaType>{};
            auto functor = UnaryOpImpl<CudaType, decltype(op)>{op};
            Elementwise<const T, T>(functor, x_cast, out);
        });
    }
};

CHAINERX_REGISTER_OP_CUDA(ArccoshOp, CudaArccoshOp);

}  // namespace
}  // namespace cuda
}  // namespace chainerx
